
#include <hip/hip_runtime.h>
#include <stdio.h>

long cantidadIntervalos = 1000000000;
long ttl_threads = 256*16;
int blockSize = 256;

float baseIntervalo = 1.0 / cantidadIntervalos;

__global__ void calc_pi(float *tmp_storage, long cantidadIntervalos, long ttl_threads, float baseIntervalo){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= ttl_threads) return; // no computation needed in this one
    
    int stride = blockDim.x * gridDim.x;
    float loc_acum=0, fdx, x;
    
    for (long i = index; i < cantidadIntervalos; i+=stride) {
        x = (i+0.5)*baseIntervalo;
        fdx = 4 / (1 + x * x);
        loc_acum += fdx;
    }
    loc_acum *= baseIntervalo;
    tmp_storage[index] = loc_acum;
}


int main() {

    int size = ttl_threads * sizeof(float);
    float* h_tmp_storage = (float*)malloc(size);
    float* d_tmp_storage;
    hipMalloc((void**)&d_tmp_storage, size);

    memset(h_tmp_storage, 0.0, size);
    hipMemcpy(d_tmp_storage, h_tmp_storage, size, hipMemcpyHostToDevice);

    int numberBlocks = (ttl_threads + blockSize - 1) / blockSize;
    calc_pi <<<numberBlocks, blockSize>>> (d_tmp_storage, cantidadIntervalos, ttl_threads, baseIntervalo);
    
	hipDeviceSynchronize();

    hipMemcpy(h_tmp_storage, d_tmp_storage, size, hipMemcpyDeviceToHost);
    
    float acum = 0;
    for(int i = 0; i < ttl_threads; i++) acum += h_tmp_storage[i];

    printf("Result = %20.18lf\n", acum);
    return 0;
}